#include "hip/hip_runtime.h" /* CUDA runtime API */
#include "cstdio"
#include "math.h"
#include <sys/time.h>


/*****************************************************************************/
/***                      local definition                        ************/
/*****************************************************************************/
#define    MAX_NUMBER            (1000000000)
#define    NEEDED_PRIME_NUM      (5)

/* Make sure the following definition satisfy the condition:
** (CPU_CALC_END * CPU_CALC_END) > MAX_NUMBER
** CPU runs the sieve arithmetic for the range [2, CPU_CALC_END)
** and GPU runs the remain part [CPU_CALC_END, MAX_NUMBER]    */
#define    CPU_CALC_END          (32000)


typedef struct
{
    int smallPrime;
    int largePrime;
    int distance;
} primeInfo;

/*****************************************************************************/
/***                    Static Databases/Variables                       *****/
/*****************************************************************************/
/* The biggest NEEDED_PRIME_NUM distances between continuous prime number in
** sorted list. The largest distance will be saved at the first one
** primeList[0]. Here, one more item is defined for simplify the calculation
** in loop.                                                                  */
primeInfo primeList[NEEDED_PRIME_NUM+1];
/* The number of found prime number. Range: 0 ~ NEEDED_PRIME_NUM */
int foundPrimeNum;


/* Function insertNewDistance(int distance, int smallerPrime, int largePrime)
*******************************************************************************
* Function description: getPrimeCUDA() is used to insert the found new large
* distance.
*
* Inputs:
*   distance: the new distance between smaller prime and larger prime
*   smallerPrime: small prime
*   largerPrime: large prime
*
* Output:
*   Save the data to array primeList[] and update variable 'foundPrimeNum'
*
* Return:
*   None
*
******************************************************************************/
void insertNewDistance(int distance, int smallPrime, int largePrime)
{
    int j;

    for (j=foundPrimeNum; j>=0; j--)
    {
        /* Save the new result to the sorted place */
        /* Note: 6 items are defined in array primeList to avoid overrun */
        if ( (0 == j) || (distance <= primeList[j - 1].distance))
        {
            primeList[j].smallPrime = smallPrime;
            primeList[j].largePrime = largePrime;
            primeList[j].distance = distance;
            break;
        }
        else if (NEEDED_PRIME_NUM != j)
        {
            /* Move the item */
            primeList[j].smallPrime = primeList[j-1].smallPrime;
            primeList[j].largePrime = primeList[j-1].largePrime;
            primeList[j].distance = primeList[j-1].distance;
        }
    }

    /* Maximum 5 largest distances are kept. Update the number of distances */
    if (foundPrimeNum < NEEDED_PRIME_NUM)
    {
        foundPrimeNum++;
    }
}

/* Function getPrimeCUDA()
*******************************************************************************
* Function description: getPrimeCUDA() is used to find out all the primes.
*
* Inputs:
*   dev: the pointer to whole buffer in device [0, 1000000000]
*   prm: the array where the prime numbers in [2, CPU_CALC_END] found were save.
*   limit: the prime numbers found by CPU (cuda limit)
*
* Output:
*   The result is saved in the memory through pointer dev
*
* Return:
*   None
*
******************************************************************************/
__global__  void getPrimeCUDA(unsigned char* dev, int* prm, int limit)
{
    int j;
    int testPrime;
    int x;

    x = blockIdx.x * blockDim.x + threadIdx.x;

    if (x < limit)
	{
        testPrime = prm[x];
        for (j=testPrime+testPrime; j<MAX_NUMBER; j+=testPrime)
        {
            dev[j]=0;
		}
	}
}


int main()
{
    unsigned char* sieve;
    unsigned char* devA;
    /* Save the found prime and pass them to cuda. The length is estimated: 1-1/2-1/3 = 1/6 */
	int primeByCPU[CPU_CALC_END/6];
    int* devPrimes;

	int foundByCPU = 0;
    int i;
    int j;
    int blockSize, nBlocks;
    int totalSize;
    struct timeval  startTime; /* Record the start time */
    struct timeval  currentTime;  /* Record the current time */

    int recSmallDist = 0;           /* Smallest distance in the 5 recorded distance */
    int lastPrime = 2;               /* Record of last prime */
    int currDistance;

    /* Verify setting for searching primes between [2, MAX_NUMBER]. */
    if ((CPU_CALC_END * CPU_CALC_END) <= MAX_NUMBER)
    {
       printf("The CPU_CALC_END * CPU_CALC_END is too small.\n", CPU_CALC_END);

       j = 2;
       while ((j*j) < MAX_NUMBER)
       {
          j++;
       }
       printf("Please change definition of  from (%d) to value no less than (%d).\n", CPU_CALC_END, j);
       return 0;
    }

    totalSize = sizeof(unsigned char)*MAX_NUMBER;
    sieve = (unsigned char*)malloc(totalSize);

    /* allocate arrays on device */
    hipMalloc((void **) &devA, totalSize);

    for (i=2; i<MAX_NUMBER; i++)
    {
        sieve[i]=1; //initialize
    }

    gettimeofday(&startTime, NULL);

    for (i=2; i<CPU_CALC_END; i++)
    {
        if(0 == sieve[i])
        {
            continue;
        }

        for (j=i+i;j<CPU_CALC_END;j=j+i)
        {
            sieve[j]=0;
        }

        primeByCPU[foundByCPU++] = i;

        currDistance = i - lastPrime;

        /* The current distance is larger than the smallest record distance. Save it. */
        if (currDistance >= recSmallDist)
        {
            insertNewDistance(currDistance, lastPrime, i);

            /* Update the current smallest distance */
            recSmallDist = primeList[foundPrimeNum - 1].distance;
        }

        lastPrime = i;
    }

    /* allocate arrays on device */
    hipMalloc((void **) &devPrimes, foundByCPU * sizeof(int));

    /* copy arrays to device memory (synchronous) */
    hipMemcpy(devA, sieve, totalSize, hipMemcpyHostToDevice);

    /* copy arrays to device memory (synchronous) */
    hipMemcpy(devPrimes, primeByCPU, foundByCPU * sizeof(int), hipMemcpyHostToDevice);
    /* guarantee synchronization */
    hipDeviceSynchronize();

    blockSize = 512;
    nBlocks = foundByCPU / blockSize;
	
	if (0 !=(foundByCPU % blockSize))
	{
	    nBlocks++;
	}

    /* execute kernel (asynchronous!) */
    getPrimeCUDA<<<nBlocks, blockSize>>>(devA, devPrimes, foundByCPU);

    /* retrieve results from device (synchronous) */
    hipMemcpy(&sieve[CPU_CALC_END], &devA[CPU_CALC_END], totalSize-CPU_CALC_END, hipMemcpyDeviceToHost);
	hipMemcpy(primeByCPU, devPrimes, foundByCPU * sizeof(int), hipMemcpyDeviceToHost);

    /* guarantee synchronization */
    hipDeviceSynchronize();
	
    for (i=CPU_CALC_END; i<MAX_NUMBER; i++)
    {
        if(sieve[i]==0)
        {
            continue;
        }

        currDistance = i - lastPrime;

        /* The current distance is larger than the smallest record distance. Save it. */
        if (currDistance >= recSmallDist)
        {
            insertNewDistance(currDistance, lastPrime, i);

            /* Update the current smallest distance */
            recSmallDist = primeList[foundPrimeNum - 1].distance;
        }

        lastPrime = i;
    }

    gettimeofday(&currentTime, NULL);
    printf("Largest prime number is %d. Total prime blocks (%d)\n", lastPrime, nBlocks);
    printf("Now, print the %d biggest distances between two continue prime numbers.\n", NEEDED_PRIME_NUM);
    for(i=0;i<NEEDED_PRIME_NUM;i++)
    {
        printf("Between continue prime number (%d) and (%d), the distance is (%d). \n", primeList[i].smallPrime, primeList[i].largePrime, primeList[i].distance);
    }
    printf ("Total time taken by CPU:  %f seconds\n",
             (double) (currentTime.tv_usec - startTime.tv_usec) / 1000000 +
             (double) (currentTime.tv_sec - startTime.tv_sec));
    free(sieve);

    return 0;
}
